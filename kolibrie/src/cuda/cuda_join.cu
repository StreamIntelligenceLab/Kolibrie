// hash_join.cu
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

#ifdef _WIN32
#define EXPORT_SYMBOL __declspec(dllexport)
#else
#define EXPORT_SYMBOL __attribute__((visibility("default")))
#endif

extern "C" {

// CUDA kernel to perform hash join
__global__ void hash_join_kernel(
    const uint32_t* d_subjects,
    const uint32_t* d_predicates,
    const uint32_t* d_objects,
    const uint32_t* d_predicate_filter,
    const uint32_t* d_literal_filter,
    uint32_t* d_indices,
    uint32_t num_triples,
    uint32_t* d_result_count)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_triples) {
        uint32_t predicate = d_predicates[idx];

        // Apply predicate filter
        if (predicate != *d_predicate_filter) {
            return;
        }

        // Apply literal filter if provided
        if (d_literal_filter != NULL) {
            uint32_t object = d_objects[idx];
            if (object != *d_literal_filter) {
                return;
            }
        }

        // Atomically store the index of the matching triple
        uint32_t pos = atomicAdd(d_result_count, 1);
        d_indices[pos] = idx;
    }
}

// Host function to perform hash join using CUDA
EXPORT_SYMBOL void perform_hash_join_cuda(
    const uint32_t* h_subjects,
    const uint32_t* h_predicates,
    const uint32_t* h_objects,
    uint32_t num_triples,
    uint32_t predicate_filter,
    uint32_t* literal_filter, // Pass NULL if no filter
    uint32_t** h_indices,
    uint32_t* h_result_count)
{
    uint32_t *d_subjects, *d_predicates, *d_objects;
    uint32_t *d_indices, *d_predicate_filter, *d_literal_filter;
    uint32_t *d_result_count;

    // Allocate device memory
    hipMalloc((void**)&d_subjects, num_triples * sizeof(uint32_t));
    hipMalloc((void**)&d_predicates, num_triples * sizeof(uint32_t));
    hipMalloc((void**)&d_objects, num_triples * sizeof(uint32_t));
    hipMalloc((void**)&d_indices, num_triples * sizeof(uint32_t));
    hipMalloc((void**)&d_predicate_filter, sizeof(uint32_t));
    hipMalloc((void**)&d_result_count, sizeof(uint32_t));
    if (literal_filter != NULL) {
        hipMalloc((void**)&d_literal_filter, sizeof(uint32_t));
    } else {
        d_literal_filter = NULL;
    }

    // Copy data to device
    hipMemcpy(d_subjects, h_subjects, num_triples * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_predicates, h_predicates, num_triples * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_objects, h_objects, num_triples * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_predicate_filter, &predicate_filter, sizeof(uint32_t), hipMemcpyHostToDevice);
    uint32_t zero = 0;
    hipMemcpy(d_result_count, &zero, sizeof(uint32_t), hipMemcpyHostToDevice);
    if (literal_filter != NULL) {
        hipMemcpy(d_literal_filter, literal_filter, sizeof(uint32_t), hipMemcpyHostToDevice);
    }

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_triples + threadsPerBlock - 1) / threadsPerBlock;
    hash_join_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_subjects,
        d_predicates,
        d_objects,
        d_predicate_filter,
        d_literal_filter,
        d_indices,
        num_triples,
        d_result_count);

    // Copy result count back to host
    hipMemcpy(h_result_count, d_result_count, sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Allocate host memory for indices
    *h_indices = (uint32_t*)malloc(*h_result_count * sizeof(uint32_t));

    // Copy indices back to host
    hipMemcpy(*h_indices, d_indices, *h_result_count * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_subjects);
    hipFree(d_predicates);
    hipFree(d_objects);
    hipFree(d_indices);
    hipFree(d_predicate_filter);
    hipFree(d_result_count);
    if (d_literal_filter != NULL) {
        hipFree(d_literal_filter);
    }
}
}
