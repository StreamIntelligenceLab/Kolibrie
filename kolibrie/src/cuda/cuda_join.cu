/*
 * Copyright © 2024 Volodymyr Kadzhaia
 * Copyright © 2024 Pieter Bonte
 * KU Leuven — Stream Intelligence Lab, Belgium
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this file,
 * you can obtain one at https://mozilla.org/MPL/2.0/.
 */

// hash_join.cu
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

#ifdef _WIN32
#define EXPORT_SYMBOL __declspec(dllexport)
#else
#define EXPORT_SYMBOL __attribute__((visibility("default")))
#endif

extern "C" {

// CUDA kernel to perform hash join
__global__ void hash_join_kernel(
    const uint32_t* __restrict__ d_subjects,
    const uint32_t* __restrict__ d_predicates,
    const uint32_t* __restrict__ d_objects,
    uint32_t predicate_filter,
    uint32_t* d_indices,
    uint32_t num_triples,
    uint32_t* d_result_count)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_triples) {
        if (d_predicates[idx] != predicate_filter) {
            return; // Skip non-matching predicate
        }

        // Atomically store the index of the matching triple
        uint32_t pos = atomicAdd(d_result_count, 1);
        d_indices[pos] = idx;
    }
}

// Host function to perform hash join using CUDA
EXPORT_SYMBOL void perform_hash_join_cuda(
    const uint32_t* h_subjects,
    const uint32_t* h_predicates,
    const uint32_t* h_objects,
    uint32_t num_triples,
    uint32_t predicate_filter,
    uint32_t* literal_filter, // Pass NULL if no filter
    uint32_t** h_indices,
    uint32_t* h_result_count)
{
    uint32_t *d_subjects, *d_predicates, *d_objects;
    uint32_t *d_indices, *d_result_count;

    // Allocate device memory
    hipMalloc((void**)&d_subjects, num_triples * sizeof(uint32_t));
    hipMalloc((void**)&d_predicates, num_triples * sizeof(uint32_t));
    hipMalloc((void**)&d_objects, num_triples * sizeof(uint32_t));
    hipMalloc((void**)&d_indices, num_triples * sizeof(uint32_t));
    hipMalloc((void**)&d_result_count, sizeof(uint32_t));

    // Copy data to device
    hipMemcpy(d_subjects, h_subjects, num_triples * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_predicates, h_predicates, num_triples * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_objects, h_objects, num_triples * sizeof(uint32_t), hipMemcpyHostToDevice);

    uint32_t zero = 0;
    hipMemcpy(d_result_count, &zero, sizeof(uint32_t), hipMemcpyHostToDevice);

    // Query device properties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // Configure kernel launch parameters
    int threadsPerBlock = deviceProp.maxThreadsPerBlock; // Maximum threads per block
    int blocksPerGrid = (num_triples + threadsPerBlock - 1) / threadsPerBlock;

    // Ensure we don't exceed the maximum number of blocks
    int maxBlocks = deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor / threadsPerBlock;
    if (blocksPerGrid > maxBlocks) {
        blocksPerGrid = maxBlocks;
    }

    // Launch kernel
    hash_join_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_subjects,
        d_predicates,
        d_objects,
        predicate_filter, // Pass scalar value
        d_indices,
        num_triples,
        d_result_count);

    // Synchronize device
    hipDeviceSynchronize();

    // Copy result count back to host
    hipMemcpy(h_result_count, d_result_count, sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Allocate host memory for indices
    *h_indices = (uint32_t*)malloc(*h_result_count * sizeof(uint32_t));

    // Copy indices back to host
    hipMemcpy(*h_indices, d_indices, *h_result_count * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_subjects);
    hipFree(d_predicates);
    hipFree(d_objects);
    hipFree(d_indices);
    hipFree(d_result_count);
}
}
